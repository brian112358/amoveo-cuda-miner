#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include <stdbool.h>
#include <stdint.h>
#include <stdlib.h>

extern "C" {
	#include "sha256.h"
	#include "utils.h"
}

#define DATASIZE 55

__global__ void kernel_sha256(BYTE *data, unsigned int* difficulty, Nonce_result *nr,unsigned int *multiplier);
__device__ WORD hash2int(BYTE h[32]);

inline void gpuAssert(hipError_t code, char *file, int line, bool abort)
{
    if (code != hipSuccess)
    {
	fprintf(stderr,"CUDA_SAFE_CALL: %s %s %d\n", hipGetErrorString(code), file, line);
	if (abort) exit(code);
    }
}

#define CUDA_SAFE_CALL(ans) { gpuAssert((ans), __FILE__, __LINE__, true); }

extern "C" bool amoveo_mine_gpu(BYTE nonce[23],unsigned int difficulty,BYTE data[55],unsigned int GDIM, unsigned int BDIM, unsigned int multiplier, unsigned int nonceRounds,double *numHashes)
{   
    //Initialize Cuda Grid variables
    dim3 DimGrid(GDIM,GDIM);
    dim3 DimBlock(BDIM,1);
    
    //Used to store a nonce if a block is mined
    Nonce_result h_nr;
    initialize_nonce_result(&h_nr);
    
    //Allocate space on Global Memory
    BYTE *d_data;
    unsigned int *d_difficulty;
    Nonce_result *d_nr;
    unsigned int *d_multiplier;
    
    CUDA_SAFE_CALL(hipMalloc((void **)&d_data, DATASIZE*sizeof(BYTE)));
    CUDA_SAFE_CALL(hipMalloc((void **)&d_difficulty, sizeof(unsigned int)));
    CUDA_SAFE_CALL(hipMalloc((void **)&d_nr, sizeof(Nonce_result)));
    CUDA_SAFE_CALL(hipMalloc((void **)&d_multiplier, sizeof(unsigned int)));
  
    //Copy data to device
    CUDA_SAFE_CALL(hipMemcpy(d_data, (void *) data, DATASIZE*sizeof(BYTE), hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpy(d_difficulty, (void *) &difficulty, sizeof(unsigned int), hipMemcpyHostToDevice));       
    CUDA_SAFE_CALL(hipMemcpy(d_nr, (void *) &h_nr, sizeof(Nonce_result), hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpy(d_multiplier, (void *) &multiplier, sizeof(unsigned int), hipMemcpyHostToDevice));

    kernel_sha256<<<DimGrid, DimBlock>>>(d_data,d_difficulty,d_nr,d_multiplier);
  
    //Copy nonce result back to host
    CUDA_SAFE_CALL(hipMemcpy((void *) &h_nr, d_nr, sizeof(Nonce_result), hipMemcpyDeviceToHost));

    hipDeviceSynchronize();

    //Free memory on device
    CUDA_SAFE_CALL(hipFree(d_data));
    CUDA_SAFE_CALL(hipFree(d_difficulty));
    CUDA_SAFE_CALL(hipFree(d_nr));
    CUDA_SAFE_CALL(hipFree(d_multiplier));

    //Copy nonce if found
    if(h_nr.nonce_found){
	for(int i=32; i<55;i++)
	    nonce[i-32]=data[i];	
	for(int i=0; i<sizeof(int64_t); i++)
	    nonce[i] = ((BYTE*)(&h_nr.nonce))[i];
    }
    
    *numHashes = ((double)GDIM)*((double)GDIM)*((double)BDIM);
    return h_nr.nonce_found;
}

//Amoveo's hash2int function to calculate difficulty
__device__ WORD hash2int(BYTE h[32]) {
  WORD x = 0;
  WORD z = 0;
  for (int i = 0; i < 31; i++) {
    if (h[i] == 0) {
      x += 8;
      continue;
    } else if (h[i] < 2) {
      x += 7;
      z = h[i+1];
    } else if (h[i] < 4) {
      x += 6;
      z = (h[i+1] / 2) + ((h[i] % 2) * 128);
    } else if (h[i] < 8) {
      x += 5;
      z = (h[i+1] / 4) + ((h[i] % 4) * 64);
    } else if (h[i] < 16) {
      x += 4;
      z = (h[i+1] / 8) + ((h[i] % 8) * 32);
    } else if (h[i] < 32) {
      x += 3;
      z = (h[i+1] / 16) + ((h[i] % 16) * 16);
    } else if (h[i] < 64) {
      x += 2;
      z = (h[i+1] / 32) + ((h[i] % 32) * 8);
    } else if (h[i] < 128) {
      x += 1;
      z = (h[i+1] / 64) + ((h[i] % 64) * 4);
    } else {
      z = (h[i+1] / 128) + ((h[i] % 128) * 2);
    }
    break;
  }
  WORD y[2];
  y[0] = x;
  y[1] = z;
  return 256*y[0]+y[1];
}

//Constants for SHA-256
__device__ static const WORD k[64] = {
  0x428a2f98,0x71374491,0xb5c0fbcf,0xe9b5dba5,0x3956c25b,0x59f111f1,0x923f82a4,0xab1c5ed5,
  0xd807aa98,0x12835b01,0x243185be,0x550c7dc3,0x72be5d74,0x80deb1fe,0x9bdc06a7,0xc19bf174,
  0xe49b69c1,0xefbe4786,0x0fc19dc6,0x240ca1cc,0x2de92c6f,0x4a7484aa,0x5cb0a9dc,0x76f988da,
  0x983e5152,0xa831c66d,0xb00327c8,0xbf597fc7,0xc6e00bf3,0xd5a79147,0x06ca6351,0x14292967,
  0x27b70a85,0x2e1b2138,0x4d2c6dfc,0x53380d13,0x650a7354,0x766a0abb,0x81c2c92e,0x92722c85,
  0xa2bfe8a1,0xa81a664b,0xc24b8b70,0xc76c51a3,0xd192e819,0xd6990624,0xf40e3585,0x106aa070,
  0x19a4c116,0x1e376c08,0x2748774c,0x34b0bcb5,0x391c0cb3,0x4ed8aa4a,0x5b9cca4f,0x682e6ff3,
  0x748f82ee,0x78a5636f,0x84c87814,0x8cc70208,0x90befffa,0xa4506ceb,0xbef9a3f7,0xc67178f2
};

//SHA-256 functions taken from Brad Conte's implementation
//https://github.com/B-Con/crypto-algorithms/blob/master/sha256.c
__device__ void d_sha256_transform(SHA256_CTX *ctx, const BYTE data[])
{
  WORD a, b, c, d, e, f, g, h, i, j, t1, t2, m[64];

  for (i = 0, j = 0; i < 16; ++i, j += 4)
    m[i] = (data[j] << 24) | (data[j + 1] << 16) | (data[j + 2] << 8) | (data[j + 3]);
  for ( ; i < 64; ++i)
    m[i] = SIG1(m[i - 2]) + m[i - 7] + SIG0(m[i - 15]) + m[i - 16];

  a = ctx->state[0];
  b = ctx->state[1];
  c = ctx->state[2];
  d = ctx->state[3];
  e = ctx->state[4];
  f = ctx->state[5];
  g = ctx->state[6];
  h = ctx->state[7];

  for (i = 0; i < 64; ++i) {
    t1 = h + EP1(e) + CH(e,f,g) + k[i] + m[i];
    t2 = EP0(a) + MAJ(a,b,c);
    h = g;
    g = f;
    f = e;
    e = d + t1;
    d = c;
    c = b;
    b = a;
    a = t1 + t2;
  }

  ctx->state[0] += a;
  ctx->state[1] += b;
  ctx->state[2] += c;
  ctx->state[3] += d;
  ctx->state[4] += e;
  ctx->state[5] += f;
  ctx->state[6] += g;
  ctx->state[7] += h;
}

__device__ void d_sha256_init(SHA256_CTX *ctx)
{
  ctx->datalen = 0;
  ctx->bitlen = 0;
  ctx->state[0] = 0x6a09e667;
  ctx->state[1] = 0xbb67ae85;
  ctx->state[2] = 0x3c6ef372;
  ctx->state[3] = 0xa54ff53a;
  ctx->state[4] = 0x510e527f;
  ctx->state[5] = 0x9b05688c;
  ctx->state[6] = 0x1f83d9ab;
  ctx->state[7] = 0x5be0cd19;
}

__device__ void d_sha256_update(SHA256_CTX *ctx, const BYTE data[], size_t len)
{
  WORD i;

  for (i = 0; i < len; ++i) {
    ctx->data[ctx->datalen] = data[i];
    ctx->datalen++;
    if (ctx->datalen == 64) {
      d_sha256_transform(ctx, ctx->data);
      ctx->bitlen += 512;
      ctx->datalen = 0;
    }
  }
}

__device__ void d_sha256_final(SHA256_CTX *ctx, BYTE hash[])
{
  WORD i;

  i = ctx->datalen;

  // Pad whatever data is left in the buffer.
  ctx->data[i++] = 0x80;
  while (i < 56)
      ctx->data[i++] = 0x00;

  // Append to the padding the total message's length in bits and transform.
  ctx->bitlen += ctx->datalen * 8;
  ctx->data[63] = ctx->bitlen;
  ctx->data[62] = ctx->bitlen >> 8;
  ctx->data[61] = ctx->bitlen >> 16;
  ctx->data[60] = ctx->bitlen >> 24;
  ctx->data[59] = ctx->bitlen >> 32;
  ctx->data[58] = ctx->bitlen >> 40;
  ctx->data[57] = ctx->bitlen >> 48;
  ctx->data[56] = ctx->bitlen >> 56;

  d_sha256_transform(ctx, ctx->data);

  // Since this implementation uses little endian byte ordering and SHA uses big endian,
  // reverse all the bytes when copying the final state to the output hash.
  for (i = 0; i < 4; ++i) {
    hash[i]      = (ctx->state[0] >> (24 - i * 8)) & 0x000000ff;
    hash[i + 4]  = (ctx->state[1] >> (24 - i * 8)) & 0x000000ff;
    hash[i + 8]  = (ctx->state[2] >> (24 - i * 8)) & 0x000000ff;
    hash[i + 12] = (ctx->state[3] >> (24 - i * 8)) & 0x000000ff;
    hash[i + 16] = (ctx->state[4] >> (24 - i * 8)) & 0x000000ff;
    hash[i + 20] = (ctx->state[5] >> (24 - i * 8)) & 0x000000ff;
    hash[i + 24] = (ctx->state[6] >> (24 - i * 8)) & 0x000000ff;
    hash[i + 28] = (ctx->state[7] >> (24 - i * 8)) & 0x000000ff;
  }
}

#define NONCE_VAL (gridDim.x*blockDim.x*blockIdx.y + blockDim.x*blockIdx.x + threadIdx.x)

__global__ void kernel_sha256(BYTE *data, unsigned int *difficulty, Nonce_result *nr, unsigned int* multiplier) {

  if(nr->nonce_found) return;
  int i;
  int64_t nonce = gridDim.x*gridDim.x;
  nonce *= blockDim.x;
  nonce *= *multiplier;
  nonce += NONCE_VAL;

  BYTE* byte_nonce = (BYTE *)&nonce;

  BYTE l_data[55];
  for(i=0;i<55;i++)
      l_data[i] = data[i];
  for(i=0;i<sizeof(int64_t);i++)
      l_data[32+i] = byte_nonce[i];
  
  SHA256_CTX ctx;
  d_sha256_init(&ctx);
  d_sha256_update(&ctx,l_data,55);

  BYTE hash[32];
  d_sha256_final(&ctx,hash);
  
  int work = hash2int(hash);
  if( work > *difficulty)
  {
      nr->nonce_found = true;
      nr->nonce = nonce;
  }
}

